#include "hip/hip_runtime.h"
#include "iostream"
#include "vector"
#include "hip/hip_runtime.h"
#include "hipblas.h"
using namespace std;

#define batchSize 96
#define M 197
#define N 768
#define thread 1024

// cuda API error checking
static void CUDA_CHECK(hipError_t err){
    if(err != hipSuccess){
        printf("CUDA error %d in %s at line %d\n", err,__FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }
}
// cuBLAS API error checking
static void CUBLAS_CHECK(hipblasStatus_t err){
    if(err != HIPBLAS_STATUS_SUCCESS){
        printf("CUBLAS error %d in %s at line %d\n", err, __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }
}

vector<vector<float>> tensor(batchSize, vector<float>(M * N));
vector<vector<float>> tensor_copy(batchSize, vector<float>(M * N));

vector<vector<float>> w1(batchSize, vector<float>(768 * 2304));
vector<vector<float>> b1(batchSize, vector<float>(197 * 2304));
vector<vector<float>> w2(batchSize, vector<float>(768 * 3072));
vector<vector<float>> b2(batchSize, vector<float>(197 * 3072));
vector<vector<float>> w3(batchSize, vector<float>(3072 * 768));
vector<vector<float>> b3(batchSize, vector<float>(197 * 768));

vector<vector<float>> Q(batchSize * 8, vector<float>(197 * 96));
vector<vector<float>> K(batchSize * 8, vector<float>(197 * 96));
vector<vector<float>> V(batchSize * 8, vector<float>(197 * 96));
vector<vector<float>> QK(batchSize * 8, vector<float>(197 * 197));


void tensor_init(int m, int n){
    for (int b = 0; b < batchSize; b++)
        for (int i = 0; i < m; i++)
            for (int j = 0; j < n; j++)
                tensor[b][j * m + i] = (float)(rand() % 101) / 101;
}

void linear_init(){
    const int m = M;
    const int n = N;
    const int k = 2304;
    const int t = 3072;

    for (int b = 0; b < batchSize; b++)
        for (int i = 0; i < n; i++)
            for (int j = 0; j < k; j++)
                w1[b][j * n + i] = (float)(rand() % 101) / 101;
    for (int b = 0; b < batchSize; b++)
        for (int i = 0; i < m; i++)
            for (int j = 0; j < k; j++)
                b1[b][j * m + i] = (float)(rand() % 101) / 101;
    for (int b = 0; b < batchSize; b++)
        for (int i = 0; i < n; i++)
            for (int j = 0; j < t; j++)
                w2[b][j * n + i] = (float)(rand() % 101) / 101;
    for (int b = 0; b < batchSize; b++)
        for (int i = 0; i < m; i++)
            for (int j = 0; j < t; j++)
                b2[b][j * m + i] = (float)(rand() % 101) / 101;
    for (int b = 0; b < batchSize; b++)
        for (int i = 0; i < t; i++)
            for (int j = 0; j < n; j++)
                w3[b][j * t + i] = (float)(rand() % 101) / 101;
    for (int b = 0; b < batchSize; b++)
        for (int i = 0; i < m; i++)
            for (int j = 0; j < n; j++)
                b3[b][j * m + i] = (float)(rand() % 101) / 101;

}

void qkv_init(int m, int n){
    const int m_batchSize = batchSize * 8;
    for (int b = 0; b < m_batchSize; b++){
        for (int i = 0; i < m; i++){
            for (int j = 0; j < n; j++){
                Q[b][j * m + i] = (float)(rand() % 101) / 101;
                K[b][j * m + i] = (float)(rand() % 101) / 101;
                V[b][j * m + i] = (float)(rand() % 101) / 101;
            }
        }
    }
}

float Att_Linear(int input_dim, int output_dim){
    const int m = M;
    const int n = output_dim;
    const int k = input_dim;
    const int lda = m;
    const int ldb = k;
    const int ldc = m;

    const float alpha = 1.0;
    const float beta = 1.0;

    hipblasHandle_t cublasH = NULL;
    hipStream_t stream = NULL;

    float** d_A_array = nullptr;
    float** d_B_array = nullptr;
    float** d_C_array = nullptr;

    vector<float*> d_A(batchSize, nullptr);
    vector<float*> d_B(batchSize, nullptr);
    vector<float*> d_C(batchSize, nullptr);

    hipblasOperation_t transa = HIPBLAS_OP_N;
    hipblasOperation_t transb = HIPBLAS_OP_N;

    /* step 1: create cublas handle, bind a stream */
    CUBLAS_CHECK(hipblasCreate(&cublasH));
    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUBLAS_CHECK(hipblasSetStream(cublasH, stream));

    /* step 2: copy data to device */
    for (int i = 0; i < batchSize; i++){
        CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_A[i]), sizeof(float) * tensor[i].size()));
        CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_B[i]), sizeof(float) * w1[i].size()));
        CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_C[i]), sizeof(float) * b1[i].size()));
    }

    CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_A_array), sizeof(float*) * batchSize));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_B_array), sizeof(float*) * batchSize));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_C_array), sizeof(float*) * batchSize));

    for (int i = 0; i < batchSize; i++){
        CUDA_CHECK(hipMemcpyAsync(d_A[i], tensor[i].data(), sizeof(float) * tensor[i].size(), hipMemcpyHostToDevice, stream));
        CUDA_CHECK(hipMemcpyAsync(d_B[i], w1[i].data(), sizeof(float) * w1[i].size(), hipMemcpyHostToDevice, stream));
        CUDA_CHECK(hipMemcpyAsync(d_C[i], b1[i].data(), sizeof(float) * b1[i].size(), hipMemcpyHostToDevice, stream));
    }

    CUDA_CHECK(hipMemcpyAsync(d_A_array, d_A.data(), sizeof(float*) * batchSize, hipMemcpyHostToDevice, stream));
    CUDA_CHECK(hipMemcpyAsync(d_B_array, d_B.data(), sizeof(float*) * batchSize, hipMemcpyHostToDevice, stream));
    CUDA_CHECK(hipMemcpyAsync(d_C_array, d_C.data(), sizeof(float*) * batchSize, hipMemcpyHostToDevice, stream));
   
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    /* step 3: compute */
    hipblasSgemmBatched(cublasH, transa, transb, m, n, k, &alpha, d_A_array, lda, d_B_array, ldb, &beta, d_C_array, ldc, batchSize);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    /* step 4: copy data to host */
    for (int i = 0; i < batchSize; i++)
        CUDA_CHECK(hipMemcpyAsync(b1[i].data(), d_C[i], sizeof(float) * b1[i].size(), hipMemcpyDeviceToHost, stream));

    /* free resources */
    CUDA_CHECK(hipFree(d_A_array));
    CUDA_CHECK(hipFree(d_B_array));
    CUDA_CHECK(hipFree(d_C_array));
    for (int i = 0; i < batchSize; i++){
        CUDA_CHECK(hipFree(d_A[i]));
        CUDA_CHECK(hipFree(d_B[i]));
        CUDA_CHECK(hipFree(d_C[i]));
    }

    CUBLAS_CHECK(hipblasDestroy(cublasH));
    CUDA_CHECK(hipStreamDestroy(stream));

    return elapsedTime;

}


float MatrixMul_1(){
    hipblasHandle_t cublasH = NULL;
    hipStream_t stream = NULL;

    const int m = M;
    const int n = M;
    const int k = N / 8;
    const int lda = m;
    const int ldb = n;
    const int ldc = m;
    const int m_batchSize = batchSize * 8;

    const float alpha = 1 / pow(24, -0.5);
    const float beta = 0;

    float** d_A_array = nullptr;
    float** d_B_array = nullptr;
    float** d_C_array = nullptr;

    vector<float*> d_A(m_batchSize, nullptr);
    vector<float*> d_B(m_batchSize, nullptr);
    vector<float*> d_C(m_batchSize, nullptr);

    hipblasOperation_t transa = HIPBLAS_OP_N;
    hipblasOperation_t transb = HIPBLAS_OP_T;

    /* step 1: create cublas handle, bind a stream */
    CUBLAS_CHECK(hipblasCreate(&cublasH));

    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUBLAS_CHECK(hipblasSetStream(cublasH, stream));

    /* step 2: copy data to device */
    for (int i = 0; i < m_batchSize; i++){
        CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_A[i]), sizeof(float) * Q[i].size()));
        CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_B[i]), sizeof(float) * K[i].size()));
        CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_C[i]), sizeof(float) * QK[i].size()));
    }

    CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_A_array), sizeof(float*) * batchSize));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_B_array), sizeof(float*) * batchSize));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_C_array), sizeof(float*) * batchSize));

    for (int i = 0; i < m_batchSize; i++){
        CUDA_CHECK(hipMemcpyAsync(d_A[i], Q[i].data(), sizeof(float) * Q[i].size(), hipMemcpyHostToDevice, stream));
        CUDA_CHECK(hipMemcpyAsync(d_B[i], K[i].data(), sizeof(float) * K[i].size(), hipMemcpyHostToDevice, stream));
        CUDA_CHECK(hipMemcpyAsync(d_C[i], QK[i].data(), sizeof(float) * QK[i].size(), hipMemcpyHostToDevice, stream));
    }

    CUDA_CHECK(hipMemcpyAsync(d_A_array, d_A.data(), sizeof(float*) * batchSize, hipMemcpyHostToDevice, stream));
    CUDA_CHECK(hipMemcpyAsync(d_B_array, d_B.data(), sizeof(float*) * batchSize, hipMemcpyHostToDevice, stream));
    CUDA_CHECK(hipMemcpyAsync(d_C_array, d_C.data(), sizeof(float*) * batchSize, hipMemcpyHostToDevice, stream));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    /* step 3: compute */
    hipblasSgemmBatched(cublasH, transa, transb, m, n, k, &alpha, d_A_array, lda, d_B_array, ldb, &beta, d_C_array, ldc, batchSize);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    /* step 4: copy data to host */
    for (int i = 0; i < m_batchSize; i++)
        CUDA_CHECK(hipMemcpyAsync(QK[i].data(), d_C[i], sizeof(float) * QK[i].size(), hipMemcpyDeviceToHost, stream));

    CUDA_CHECK(hipStreamSynchronize(stream));

    /* free resources */
    CUDA_CHECK(hipFree(d_A_array));
    CUDA_CHECK(hipFree(d_B_array));
    CUDA_CHECK(hipFree(d_C_array));
    for (int i = 0; i < m_batchSize; i++){
        CUDA_CHECK(hipFree(d_A[i]));
        CUDA_CHECK(hipFree(d_B[i]));
        CUDA_CHECK(hipFree(d_C[i]));
    }

    CUBLAS_CHECK(hipblasDestroy(cublasH));
    CUDA_CHECK(hipStreamDestroy(stream));

    return elapsedTime;
}

float softmax(){
    const int m = M;
    const int count = batchSize * 8 * m;
    float s[count] = {};
    int t = 0;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    for (int b = 0; b < batchSize * 8; b++){
        for (int i = 0; i < m; i++){
            for (int j = 0; j < m; j++){
                s[t]= s[t]+ exp(QK[b][j * m + i]);
            }
            t++;
        }
    }
    t = 0;
    for (int b = 0; b < batchSize * 8; b++)
        for (int i = 0; i < m; i++) {
            for (int j = 0; j < m; j++) {
                QK[b][j * m + i] = exp(QK[b][j * m + i]) / s[t];
            }
            t++;
        }
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return elapsedTime;
}

float MatrixMul_2(){
    hipblasHandle_t cublasH = NULL;
    hipStream_t stream = NULL;

    const int m = M;
    const int n = N / 8;
    const int k = m;
    const int lda = m;
    const int ldb = k; 
    const int ldc = m;
    const int m_batchSize = batchSize * 8;

    const float alpha = 1.0;
    const float beta = 0;

    float** d_A_array = nullptr;
    float** d_B_array = nullptr;
    float** d_C_array = nullptr;

    vector<float*> d_A(m_batchSize, nullptr);
    vector<float*> d_B(m_batchSize, nullptr);
    vector<float*> d_C(m_batchSize, nullptr);

    hipblasOperation_t transa = HIPBLAS_OP_N;
    hipblasOperation_t transb = HIPBLAS_OP_N; 

    /* step 1: create cublas handle, bind a stream */
    CUBLAS_CHECK(hipblasCreate(&cublasH));

    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUBLAS_CHECK(hipblasSetStream(cublasH, stream));

    /* step 2: copy data to device */
    for (int i = 0; i < m_batchSize; i++){
        CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_A[i]), sizeof(float) * QK[i].size()));
        CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_B[i]), sizeof(float) * V[i].size()));
        CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_C[i]), sizeof(float) * Q[i].size()));
    }

    CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_A_array), sizeof(float*) * batchSize));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_B_array), sizeof(float*) * batchSize));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_C_array), sizeof(float*) * batchSize));

    for (int i = 0; i < m_batchSize; i++){
        CUDA_CHECK(hipMemcpyAsync(d_A[i], QK[i].data(), sizeof(float) * QK[i].size(), hipMemcpyHostToDevice, stream));
        CUDA_CHECK(hipMemcpyAsync(d_B[i], V[i].data(), sizeof(float) * V[i].size(), hipMemcpyHostToDevice, stream));
        CUDA_CHECK(hipMemcpyAsync(d_C[i], Q[i].data(), sizeof(float) * Q[i].size(), hipMemcpyHostToDevice, stream));
    }

    CUDA_CHECK(hipMemcpyAsync(d_A_array, d_A.data(), sizeof(float*) * batchSize, hipMemcpyHostToDevice, stream));
    CUDA_CHECK(hipMemcpyAsync(d_B_array, d_B.data(), sizeof(float*) * batchSize, hipMemcpyHostToDevice, stream));
    CUDA_CHECK(hipMemcpyAsync(d_C_array, d_C.data(), sizeof(float*) * batchSize, hipMemcpyHostToDevice, stream));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    /* step 3: compute */
    hipblasSgemmBatched(cublasH, transa, transb, m, n, k, &alpha, d_A_array, lda, d_B_array, ldb, &beta, d_C_array, ldc, batchSize);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    /* step 4: copy data to host */
    for (int i = 0; i < m_batchSize; i++)
        CUDA_CHECK(hipMemcpyAsync(Q[i].data(), d_C[i], sizeof(float) * Q[i].size(), hipMemcpyDeviceToHost, stream));

    CUDA_CHECK(hipStreamSynchronize(stream));

    /* free resources */
    CUDA_CHECK(hipFree(d_A_array));
    CUDA_CHECK(hipFree(d_B_array));
    CUDA_CHECK(hipFree(d_C_array));
    for (int i = 0; i < m_batchSize; i++){
        CUDA_CHECK(hipFree(d_A[i]));
        CUDA_CHECK(hipFree(d_B[i]));
        CUDA_CHECK(hipFree(d_C[i]));
    }

    CUBLAS_CHECK(hipblasDestroy(cublasH));
    CUDA_CHECK(hipStreamDestroy(stream));
    
    return elapsedTime;
}

float MLP_Linear_1(int input_dim, int output_dim){
    const int m = M;
    const int n = output_dim;
    const int k = input_dim;
    const int lda = m;
    const int ldb = k;
    const int ldc = m;

    const float alpha = 1.0;
    const float beta = 1.0;

    hipblasHandle_t cublasH = NULL;
    hipStream_t stream = NULL;

    float** d_A_array = nullptr;
    float** d_B_array = nullptr;
    float** d_C_array = nullptr;

    vector<float*> d_A(batchSize, nullptr);
    vector<float*> d_B(batchSize, nullptr);
    vector<float*> d_C(batchSize, nullptr);

    hipblasOperation_t transa = HIPBLAS_OP_N;
    hipblasOperation_t transb = HIPBLAS_OP_N;

    /* step 1: create cublas handle, bind a stream */
    CUBLAS_CHECK(hipblasCreate(&cublasH));
    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUBLAS_CHECK(hipblasSetStream(cublasH, stream));

    /* step 2: copy data to device */
    for (int i = 0; i < batchSize; i++){
        CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_A[i]), sizeof(float) * tensor[i].size()));
        CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_B[i]), sizeof(float) * w2[i].size()));
        CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_C[i]), sizeof(float) * b2[i].size()));
    }

    CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_A_array), sizeof(float*) * batchSize));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_B_array), sizeof(float*) * batchSize));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_C_array), sizeof(float*) * batchSize));

    for (int i = 0; i < batchSize; i++){
        CUDA_CHECK(hipMemcpyAsync(d_A[i], tensor[i].data(), sizeof(float) * tensor[i].size(), hipMemcpyHostToDevice, stream));
        CUDA_CHECK(hipMemcpyAsync(d_B[i], w2[i].data(), sizeof(float) * w2[i].size(), hipMemcpyHostToDevice, stream));
        CUDA_CHECK(hipMemcpyAsync(d_C[i], b2[i].data(), sizeof(float) * b2[i].size(), hipMemcpyHostToDevice, stream));
    }

    CUDA_CHECK(hipMemcpyAsync(d_A_array, d_A.data(), sizeof(float*) * batchSize, hipMemcpyHostToDevice, stream));
    CUDA_CHECK(hipMemcpyAsync(d_B_array, d_B.data(), sizeof(float*) * batchSize, hipMemcpyHostToDevice, stream));
    CUDA_CHECK(hipMemcpyAsync(d_C_array, d_C.data(), sizeof(float*) * batchSize, hipMemcpyHostToDevice, stream));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    /* step 3: compute */
    hipblasSgemmBatched(cublasH, transa, transb, m, n, k, &alpha, d_A_array, lda, d_B_array, ldb, &beta, d_C_array, ldc, batchSize);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    /* step 4: copy data to host */
    for (int i = 0; i < batchSize; i++)
        CUDA_CHECK(hipMemcpyAsync(b2[i].data(), d_C[i], sizeof(float) * b2[i].size(), hipMemcpyDeviceToHost, stream));

    /* free resources */
    CUDA_CHECK(hipFree(d_A_array));
    CUDA_CHECK(hipFree(d_B_array));
    CUDA_CHECK(hipFree(d_C_array));
    for (int i = 0; i < batchSize; i++){
        CUDA_CHECK(hipFree(d_A[i]));
        CUDA_CHECK(hipFree(d_B[i]));
        CUDA_CHECK(hipFree(d_C[i]));
    }

    CUBLAS_CHECK(hipblasDestroy(cublasH));
    CUDA_CHECK(hipStreamDestroy(stream));

    return elapsedTime;
}

__global__ void gelu(float* x, int n){
    int id = threadIdx.x + blockDim.x * blockIdx.x;
    if (id < n)
        x[id] = 0.5 * x[id] * (1 + tanh(sqrt(2 / 3.1415926) + 0.004715 * pow(x[id], 3)));  
}

float GELU() {
    vector<float*> d_A(batchSize, nullptr);

    hipStream_t stream = NULL;
    const int block = (M * N - 0.5) / thread + 1; 
    float elapsedTime = 0.0;

    /* copy data to device */
    for (int i = 0; i < batchSize; i++){
        CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_A[i]), sizeof(float) * b2[i].size()));
        CUDA_CHECK(hipMemcpyAsync(d_A[i], b2[i].data(), sizeof(float) * QK[i].size(), hipMemcpyHostToDevice, stream));

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start);
        gelu << <block, thread >> > (d_A[i], QK[i].size());

        hipEventRecord(stop);
        hipEventSynchronize(stop);

        float tmp;
        hipEventElapsedTime(&tmp, start, stop);
        hipEventDestroy(start);
        hipEventDestroy(stop);
        elapsedTime += tmp;

        CUDA_CHECK(hipMemcpyAsync(b2[i].data(), d_A[i], sizeof(float) * b1[i].size(), hipMemcpyDeviceToHost, stream));
    }

    CUDA_CHECK(hipStreamSynchronize(stream));

    /* free resources */
    for (int i = 0; i < batchSize; i++)
        CUDA_CHECK(hipFree(d_A[i]));

    return elapsedTime;
}

float MLP_Linear_2(int input_dim, int output_dim){
    const int m = M;
    const int n = output_dim;
    const int k = input_dim;
    const int lda = m;
    const int ldb = k;
    const int ldc = m;

    const float alpha = 1.0;
    const float beta = 1.0;

    hipblasHandle_t cublasH = NULL;
    hipStream_t stream = NULL;

    float** d_A_array = nullptr;
    float** d_B_array = nullptr;
    float** d_C_array = nullptr;

    vector<float*> d_A(batchSize, nullptr);
    vector<float*> d_B(batchSize, nullptr);
    vector<float*> d_C(batchSize, nullptr);

    hipblasOperation_t transa = HIPBLAS_OP_N;
    hipblasOperation_t transb = HIPBLAS_OP_N;

    /* step 1: create cublas handle, bind a stream */
    CUBLAS_CHECK(hipblasCreate(&cublasH));
    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUBLAS_CHECK(hipblasSetStream(cublasH, stream));

    /* step 2: copy data to device */
    for (int i = 0; i < batchSize; i++){
        CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_A[i]), sizeof(float) * b2[i].size()));
        CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_B[i]), sizeof(float) * w3[i].size()));
        CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_C[i]), sizeof(float) * b3[i].size()));
    }

    CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_A_array), sizeof(float*) * batchSize));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_B_array), sizeof(float*) * batchSize));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_C_array), sizeof(float*) * batchSize));

    for (int i = 0; i < batchSize; i++){
        CUDA_CHECK(hipMemcpyAsync(d_A[i], b2[i].data(), sizeof(float) * b2[i].size(), hipMemcpyHostToDevice, stream));
        CUDA_CHECK(hipMemcpyAsync(d_B[i], w3[i].data(), sizeof(float) * w3[i].size(), hipMemcpyHostToDevice, stream));
        CUDA_CHECK(hipMemcpyAsync(d_C[i], b3[i].data(), sizeof(float) * b3[i].size(), hipMemcpyHostToDevice, stream));
    }

    CUDA_CHECK(hipMemcpyAsync(d_A_array, d_A.data(), sizeof(float*) * batchSize, hipMemcpyHostToDevice, stream));
    CUDA_CHECK(hipMemcpyAsync(d_B_array, d_B.data(), sizeof(float*) * batchSize, hipMemcpyHostToDevice, stream));
    CUDA_CHECK(hipMemcpyAsync(d_C_array, d_C.data(), sizeof(float*) * batchSize, hipMemcpyHostToDevice, stream));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    /* step 3: compute */
    hipblasSgemmBatched(cublasH, transa, transb, m, n, k, &alpha, d_A_array, lda, d_B_array, ldb, &beta, d_C_array, ldc, batchSize);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    /* step 4: copy data to host */
    for (int i = 0; i < batchSize; i++)
        CUDA_CHECK(hipMemcpyAsync(b3[i].data(), d_C[i], sizeof(float) * b3[i].size(), hipMemcpyDeviceToHost, stream));

    /* free resources */
    CUDA_CHECK(hipFree(d_A_array));
    CUDA_CHECK(hipFree(d_B_array));
    CUDA_CHECK(hipFree(d_C_array));
    for (int i = 0; i < batchSize; i++){
        CUDA_CHECK(hipFree(d_A[i]));
        CUDA_CHECK(hipFree(d_B[i]));
        CUDA_CHECK(hipFree(d_C[i]));
    }

    CUBLAS_CHECK(hipblasDestroy(cublasH));
    CUDA_CHECK(hipStreamDestroy(stream));

    return elapsedTime;
}


__global__ void var(float* x, int n, float avg){
    int id = threadIdx.x + blockDim.x * blockIdx.x;
    if (id < n)
        x[id] = pow(x[id] - avg, 2);
}

__global__ void normalize(float* x, int n, float avg, float S){
    int id = threadIdx.x + blockDim.x * blockIdx.x;
    if (id < n)
        x[id] = (x[id] - avg) / sqrt(S + 1e-5);
}

float LayerNorm(){
    vector<float*> d_A(batchSize, nullptr);
    vector<float*> d_A_copy(batchSize, nullptr);
    float sum = 0.0;
    float S = 0.0;
    hipStream_t stream = NULL;

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    const int token_size = M * N;
    const int block = (token_size - 0.5) / thread + 1;
    /* copy data to device */
    float elapsedTime = 0.0;
    for (int i = 0; i < batchSize; i++){
        sum = 0.0;
        CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_A[i]), sizeof(float) * tensor[i].size()));
        CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_A_copy[i]), sizeof(float) * tensor[i].size()));
        CUDA_CHECK(hipMemcpyAsync(d_A[i], tensor[i].data(), sizeof(float) * tensor[i].size(), hipMemcpyHostToDevice, stream));
        CUDA_CHECK(hipMemcpyAsync(d_A_copy[i], tensor[i].data(), sizeof(float) * tensor[i].size(), hipMemcpyHostToDevice, stream));

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start);

        hipblasSasum(handle, token_size, d_A[i], 1, &sum);
        var << <block, thread >> > (d_A_copy[i], token_size, sum / (token_size));
        hipblasSasum(handle, token_size, d_A_copy[i], 1, &S);
        normalize << <block, thread >> > (d_A[i], token_size, sum / (token_size), S);

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        
        float tmp;
        hipEventElapsedTime(&tmp, start, stop);
        hipEventDestroy(start);
        hipEventDestroy(stop);
        elapsedTime += tmp;
        CUDA_CHECK(hipMemcpyAsync(tensor[i].data(), d_A[i], sizeof(float) * tensor[i].size(), hipMemcpyDeviceToHost, stream));
    }

    CUDA_CHECK(hipStreamSynchronize(stream));

    /* free resources */
    for (int i = 0; i < batchSize; i++){
        CUDA_CHECK(hipFree(d_A[i]));
        CUDA_CHECK(hipFree(d_A_copy[i]));
    }

    CUBLAS_CHECK(hipblasDestroy(handle));

    return elapsedTime;
}

float Attention(){
    float time = 0.0;
    
    time += Att_Linear(N, 2304);
    time += MatrixMul_1();
    time += softmax();
    time += MatrixMul_2();

    return time;
}

float MLP(){
    float time = 0.0;

    time += MLP_Linear_1(N,3072);
    time += GELU();
    time += MLP_Linear_2(3072,N);
    tensor = b3;

    return time;
}

float Residual() {
    const int m = M;
    const int n = N;
    const int vector_size = batchSize * m * n;
    vector<float> A(vector_size);
    vector<float> B(vector_size);

    int t = 0;
    for (int b = 0; b < batchSize; b++){
        for (int i = 0; i < m; i++){
            for (int j = 0; j < n; j++){
                A[t] = tensor[b][j * m + i];
                B[t] = tensor_copy[b][j * m + i];
                t++;
            }
        }
    }

    hipblasHandle_t cublasH = NULL;
    hipStream_t stream = NULL;

    const float alpha = 1.0;
    const int incx = 1;
    const int incy = 1;

    float* d_A = nullptr;
    float* d_B = nullptr;

    /* step 1: create cublas handle, bind a stream */
    CUBLAS_CHECK(hipblasCreate(&cublasH));

    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUBLAS_CHECK(hipblasSetStream(cublasH, stream));

    /* step 2: copy data to device */
    CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_A), sizeof(float) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_B), sizeof(float) * B.size()));

    CUDA_CHECK(hipMemcpyAsync(d_A, A.data(), sizeof(float) * A.size(), hipMemcpyHostToDevice, stream));
    CUDA_CHECK(hipMemcpyAsync(d_B, B.data(), sizeof(float) * B.size(), hipMemcpyHostToDevice, stream));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    /* step 3: compute */
    CUBLAS_CHECK(hipblasSaxpy(cublasH, A.size(), &alpha, d_A, incx, d_B, incy));

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    /* step 4: copy data to host */
    CUDA_CHECK(hipMemcpyAsync(B.data(), d_B, sizeof(float) * B.size(), hipMemcpyDeviceToHost, stream));

    CUDA_CHECK(hipStreamSynchronize(stream));

    /* free resources */
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));

    CUBLAS_CHECK(hipblasDestroy(cublasH));
    CUDA_CHECK(hipStreamDestroy(stream));

    t = 0;
    for (int b = 0; b < batchSize; b++){
        for (int i = 0; i < m; i++){
            for (int j = 0; j < n; j++){
                tensor[b][j * m + i] = B[t];
                t++;
            }
        }
    }
    return elapsedTime;
}


int main(){
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    tensor_init(M, N);
    tensor_copy = tensor;
    linear_init();
    qkv_init(M, N / 8);

    for(int i = 0; i < 100; i++)
        LayerNorm();
    
    float GPU_time = 0.0;
    hipEventRecord(start);
    GPU_time += LayerNorm();
    GPU_time += Attention();
    GPU_time += Residual();
    tensor_copy = tensor;

    GPU_time += LayerNorm();
    GPU_time += MLP();
    GPU_time += Residual();

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    printf("BatchSize:%d\n", batchSize);
    printf("Execution time:%.2fms\n", elapsedTime);
    printf("GPU kernel time:%.2fms\n", GPU_time);

    return 0;
}
